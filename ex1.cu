#include "hip/hip_runtime.h"
#include "ex1.h"


#define HIST_LENGTH 256
#define IMG_SIZE IMG_WIDTH*IMG_HEIGHT

__device__ void prefix_sum(int arr[], int arr_size) {
    int threadID = threadIdx.x;
    int offset = 1;
    int last = arr[arr_size-1];
    for(int level = arr_size / 2; level > 0; level /= 2)
    {
    	if(threadID < level)
    	{
    		arr[offset * (2 * threadID + 2) - 1] += arr[offset * (2 * threadID + 1) - 1];
    	}
    	offset *= 2;
    	__syncthreads(); 
    }
    if(threadID == 0)
    {
    	arr[arr_size - 1] = 0;
    }
    for(int level = 1; level < arr_size; level *= 2)
    {
    	offset /= 2;
    	__syncthreads();
    	if(threadID < level)
    	{
    		int temp = arr[offset * (2 * threadID + 1) - 1];
    		arr[offset * (2 * threadID + 1) - 1] = arr[offset * (2 * threadID + 2) - 1];
    		arr[offset * (2 * threadID + 2) - 1] += temp;
    	}
    }
	__syncthreads(); 
    if(threadID == 0){
        for(int i=0; i<arr_size-1;i++){
        	arr[i]=arr[i+1];
        }
        arr[arr_size-1]= arr[arr_size-1]+last;
    }
	return;
}

//serial 
/*__global__ void process_image_kernel(uchar *all_in, uchar *all_out) {
    __shared__ int hist[HIST_LENGTH];

    int threadID = threadIdx.x;
    int blockSize = blockDim.x;

    if(threadID < HIST_LENGTH)
    {
    	hist[threadID] = 0;
    }

    //	Create the histogram
    for(int i = threadID; i < IMG_SIZE; i += blockSize)
    {
    	atomicAdd(&hist[all_in[i]], 1);
    }
    __syncthreads();

    //	Create the CDF
    prefix_sum(hist, HIST_LENGTH);
    __syncthreads();

    //	Create the map
    if(threadID < HIST_LENGTH)
    {
        hist[threadID] = (HIST_LENGTH / N_COLORS) * (int)(N_COLORS * (float)hist[threadID] / (IMG_WIDTH * IMG_HEIGHT));
    }
    __syncthreads();

    //	Compute the new image
    for(int i = threadID; i < IMG_SIZE; i += blockSize)
        {
        	all_out[i] = hist[all_in[i]];
        }
	return;
}*/


//block
__global__ void process_image_kernel(uchar *all_in, uchar *all_out) {
    __shared__ int hist[HIST_LENGTH];

    int threadID  =  threadIdx.x;
    int blockID   =  blockIdx.x;
    int blockSize =  blockDim.x;

    if(threadID < HIST_LENGTH)
    {
    	hist[threadID] = 0;
    }
    __syncthreads();
    //	Create the histogram
    for(int i = threadID; i < IMG_SIZE; i += blockSize)
    {
    	atomicAdd(&hist[all_in[i + blockID*IMG_SIZE]], 1);
    }
    __syncthreads();

    //	Create the CDF
    prefix_sum(hist, HIST_LENGTH);
    __syncthreads();

    //	Create the map
    if(threadID < HIST_LENGTH)
    {
        hist[threadID] = (HIST_LENGTH / N_COLORS) * (int)(N_COLORS * (float)hist[threadID] / (IMG_SIZE));
    }
    __syncthreads();

    //	Compute the new image
    for(int i = threadID; i < IMG_SIZE; i += blockSize)
        {
        	all_out[i + blockID*IMG_SIZE] = hist[all_in[i + blockID*IMG_SIZE]];
        }
	return;
}
/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    uchar *all_in, *all_out;
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    CUDA_CHECK(hipMalloc((void**)&context->all_in, IMG_SIZE*sizeof(uchar)));
    CUDA_CHECK(hipMalloc((void**)&context->all_out, IMG_SIZE*sizeof(uchar)));

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    for(int imageIdx = 0; imageIdx < N_IMAGES; imageIdx++)
    {
    	CUDA_CHECK(hipMemcpy((void*)context->all_in, (void*)(images_in + imageIdx * IMG_SIZE), IMG_SIZE*sizeof(uchar), hipMemcpyHostToDevice));
    	process_image_kernel<<<1, 1024>>>(context->all_in, context->all_out);
    	CUDA_CHECK(hipDeviceSynchronize());
    	CUDA_CHECK(hipMemcpy((void*)(images_out + imageIdx * IMG_SIZE), (void*)context->all_out, IMG_SIZE*sizeof(uchar), hipMemcpyDeviceToHost));
    }
}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    CUDA_CHECK(hipFree((void*)context->all_in));
    CUDA_CHECK(hipFree((void*)context->all_out));

    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
	uchar *all_in, *all_out;
};

/* Allocate GPU memory for all the input and output images.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    CUDA_CHECK(hipMalloc((void**)&context->all_in,  N_IMAGES*IMG_SIZE*sizeof(uchar)));
    CUDA_CHECK(hipMalloc((void**)&context->all_out, N_IMAGES*IMG_SIZE*sizeof(uchar)));

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
	CUDA_CHECK(hipMemcpy((void*)context->all_in, (void*)(images_in), N_IMAGES*IMG_SIZE*sizeof(uchar), hipMemcpyHostToDevice));
	process_image_kernel<<<N_IMAGES, 1024>>>(context->all_in, context->all_out);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemcpy((void*)(images_out), (void*)context->all_out, N_IMAGES*IMG_SIZE*sizeof(uchar), hipMemcpyDeviceToHost));
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    CUDA_CHECK(hipFree((void*)context->all_in));
    CUDA_CHECK(hipFree((void*)context->all_out));
    free(context);
}

